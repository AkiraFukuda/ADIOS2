#include "hip/hip_runtime.h"
/*
 * Distributed under the OSI-approved Apache License, Version 2.0.  See
 * accompanying file Copyright.txt for details.
 *
 * adiosCUDA.cpp
 *
 *  Created on: May 9, 2021
 *      Author: Ana Gainaru gainarua@ornl.gov
 */

#ifndef ADIOS2_HELPER_ADIOSCUDA_CU_
#define ADIOS2_HELPER_ADIOSCUDA_CU_

#include "adios2/common/ADIOSMacros.h"

#include "adiosCUDA.h"
#include "adiosCUDAReduceImpl.h"

namespace
{

template <class T>
void CUDAMinMaxImpl(const T *values, const size_t size, T &min, T &max)
{
    min = reduce<T, MinOp>(size, 1024, 64, 1, values);
    max = reduce<T, MaxOp>(size, 1024, 64, 1, values);
}
// types non supported on the device
void CUDAMinMaxImpl(const long double * /*values*/, const size_t /*size*/,
                    long double & /*min*/, long double & /*max*/)
{
}
void CUDAMinMaxImpl(const std::complex<float> * /*values*/,
                    const size_t /*size*/, std::complex<float> & /*min*/,
                    std::complex<float> & /*max*/)
{
}
void CUDAMinMaxImpl(const std::complex<double> * /*values*/,
                    const size_t /*size*/, std::complex<double> & /*min*/,
                    std::complex<double> & /*max*/)
{
}
}

template <class T>
void adios2::helper::CUDAMinMax(const T *values, const size_t size, T &min,
                                T &max)
{
    CUDAMinMaxImpl(values, size, min, max);
}

#define declare_type(T)                                                        \
    template void adios2::helper::CUDAMinMax(                                  \
        const T *values, const size_t size, T &min, T &max);
ADIOS2_FOREACH_PRIMITIVE_STDTYPE_1ARG(declare_type)
#undef declare_type

#endif /* ADIOS2_HELPER_ADIOSCUDA_CU_ */
